// Copyright (c) 2024, Tri Dao.
// Splitting the different head dimensions to different files to speed up compilation.

#include "flash_fwd_launch_template.h"

template<>
void run_mha_fwd_<cutlass::half_t, 32, 64>(Flash_fwd_params &params, hipStream_t stream) {
    run_mha_fwd_qkdim32_vdim64<cutlass::half_t>(params, stream);
}
